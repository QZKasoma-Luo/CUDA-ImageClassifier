#include "../include/cuda_utils.cuh"

void check_cuda_error(hipError_t result, const char *func, const char *file, int line)
{
    if (result != hipSuccess)
    {
        fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\" \n",
                file, line, static_cast<unsigned int>(result), hipGetErrorString(result), func);
        hipDeviceReset();
        exit(EXIT_FAILURE);
    }
}