#include "hip/hip_runtime.h"
﻿#include "../include/fc_layer.cuh"
#include "../include/cuda_utils.cuh"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <cmath>

__global__ void init_curand_states(hiprandState* state, unsigned long seed, int n) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n) {
        hiprand_init(seed, id, 0, &state[id]);
    }
}

__global__ void init_weights(float* weights, int size, float scale, hiprandState* states) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        hiprandState localState = states[idx];
        float r = sqrtf(6.0f / size);
        weights[idx] = (hiprand_uniform(&localState) * 2.0f - 1.0f) * r;
        states[idx] = localState;
    }
}

FCLayer::FCLayer(int input_size, int output_size)
    : input_size(input_size), output_size(output_size) {

    // 分配并初始化权重和偏置
    hipMalloc(&weights, input_size * output_size * sizeof(float));
    hipMalloc(&bias, output_size * sizeof(float));
    hipMalloc(&grad_weights, input_size * output_size * sizeof(float));
    hipMalloc(&grad_bias, output_size * sizeof(float));

    // 初始化 cuBLAS
    hipblasCreate(&cublas_handle);

    // 初始化 cuRAND
    hiprandCreateGenerator(&curand_gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(curand_gen, 1234ULL);

    // 初始化 cuRAND 状态
    hiprandState* d_states;
    hipMalloc(&d_states, input_size * output_size * sizeof(hiprandState));
    int blockSize = 256;
    int numBlocks = (input_size * output_size + blockSize - 1) / blockSize;
    init_curand_states << <numBlocks, blockSize >> > (d_states, 1234ULL, input_size * output_size);

    // 初始化权重
    init_weights << <numBlocks, blockSize >> > (weights, input_size * output_size, sqrtf(2.0f / input_size), d_states);

    // 初始化偏置为0
    hipMemset(bias, 0, output_size * sizeof(float));

    // 清理临时内存
    hipFree(d_states);
}

FCLayer::~FCLayer() {
    hipFree(weights);
    hipFree(bias);
    hipFree(grad_weights);
    hipFree(grad_bias);
    hipblasDestroy(cublas_handle);
    hiprandDestroyGenerator(curand_gen);
}

void FCLayer::forward(const float* input, float* output, int batch_size) {
    const float alpha = 1.0f;
    const float beta = 0.0f;

    // 计算 output = weights^T * input + bias
    hipblasSgemm(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
        output_size, batch_size, input_size,
        &alpha, weights, input_size, input, input_size,
        &beta, output, output_size);

    // 添加偏置
    for (int i = 0; i < batch_size; ++i) {
        hipMemcpy(output + i * output_size, bias, output_size * sizeof(float), hipMemcpyDeviceToDevice);
    }
}

void FCLayer::backward(const float* input, const float* grad_output, float* grad_input, int batch_size) {
    const float alpha = 1.0f;
    const float beta = 0.0f;

    // 计算输入梯度：grad_input = weights * grad_output
    hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
        input_size, batch_size, output_size,
        &alpha, weights, input_size, grad_output, output_size,
        &beta, grad_input, input_size);

    // 计算权重梯度：grad_weights = input * grad_output^T
    hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T,
        input_size, output_size, batch_size,
        &alpha, input, input_size, grad_output, output_size,
        &beta, grad_weights, input_size);

    // 计算偏置梯度
    for (int i = 0; i < batch_size; ++i) {
        hipblasSaxpy(cublas_handle, output_size, &alpha,
            grad_output + i * output_size, 1, grad_bias, 1);
    }
}

__global__ void update_params_kernel(float* params, float* grads, int size, float learning_rate) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        params[idx] -= learning_rate * grads[idx];
    }
}

void FCLayer::update_params(float learning_rate) {
    int blockSize = 256;
    int numBlocks = (input_size * output_size + blockSize - 1) / blockSize;

    // 更新权重
    update_params_kernel << <numBlocks, blockSize >> > (weights, grad_weights, input_size * output_size, learning_rate);

    // 更新偏置
    numBlocks = (output_size + blockSize - 1) / blockSize;
    update_params_kernel << <numBlocks, blockSize >> > (bias, grad_bias, output_size, learning_rate);
}